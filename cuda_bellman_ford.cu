#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <assert.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include <sys/stat.h>

#define INF 1000000

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                          \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);               \
        fprintf(stderr, "code: %d, reason: %s\n", error,                     \
                hipGetErrorString(error));                                  \
        exit(1);                                                               \
    }                                                                          \
}

typedef struct {
    int id;
} node;

typedef struct {
    int src;
    int dest;
    int weight;
} edge;

typedef struct {
    int n; // number of nodes
    int m; // number of edges
    node *nodes;
    edge *edges;
} graph;

void abort_with_error_message(const char* msg) {
    fprintf(stderr, "%s\n", msg);
    exit(1);
}

void read_graph(const char* filename, graph *G) {
    FILE *inputf = fopen(filename, "r");
    if (inputf == NULL) {
        abort_with_error_message("ERROR OCCURRED WHILE READING INPUT FILE");
    }

    fscanf(inputf, "%d %d", &G->n, &G->m);
    G->nodes = (node *)malloc(G->n * sizeof(node));
    G->edges = (edge *)malloc(G->m * sizeof(edge));

    for (int i = 0; i < G->n; i++) {
        G->nodes[i].id = i; // Initialize node ID
    }

    for (int i = 0; i < G->m; i++) {
        int src, dest, weight;
        fscanf(inputf, "%d %d %d", &src, &dest, &weight);
        G->edges[i].src = src;
        G->edges[i].dest = dest;
        G->edges[i].weight = weight;
    }

    fclose(inputf);
}

__global__ void bellman_ford_one_iter(int n, int m, edge *d_edges, int *d_dist, bool *d_has_next) {
    int global_tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (global_tid < m) {
        edge e = d_edges[global_tid];
        if (d_dist[e.src] < INF) {
            int new_dist = d_dist[e.src] + e.weight;
            if (new_dist < d_dist[e.dest]) {
                d_dist[e.dest] = new_dist;
                *d_has_next = true;
            }
        }
    }
}

void write_result(const char *filename, int *dist, int n, bool has_negative_cycle) {
    FILE *outputf = fopen(filename, "w");
    if (outputf == NULL) {
        abort_with_error_message("ERROR OCCURRED WHILE OPENING OUTPUT FILE");
    }
    if (!has_negative_cycle) {
        for (int i = 0; i < n; i++) {
            if (i == n - 1) {
                fprintf(outputf, "%d", dist[i] == INF ? INF : dist[i]); // No space after last number
            } else {
                fprintf(outputf, "%d ", dist[i] == INF ? INF : dist[i]); // Add space after each number
            }
        }
        fprintf(outputf, "\n"); // Newline at the end
    } else {
        fprintf(outputf, "FOUND NEGATIVE CYCLE!\n");
    }
    fclose(outputf);
}
void bellman_ford(graph *G, int *dist, bool *has_negative_cycle, int blocksPerGrid, int threadsPerBlock) {
    int *d_dist;
    edge *d_edges;
    bool *d_has_next, h_has_next;

    struct timeval start, end;
    double alloc_time, exec_time = 0.0, h2d_time = 0.0, d2h_time = 0.0;

    // Memory allocation/setup timing
    gettimeofday(&start, NULL);

    hipMalloc(&d_edges, G->m * sizeof(edge));
    hipMalloc(&d_dist, sizeof(int) * G->n);
    hipMalloc(&d_has_next, sizeof(bool));

    *has_negative_cycle = false;

    for (int i = 0; i < G->n; i++) {
        dist[i] = INF;
    }
    dist[0] = 0; // Assuming the source node is 0

    gettimeofday(&end, NULL);
    alloc_time = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1000000.0;


    // Host to Device copy timing
    gettimeofday(&start, NULL);
    hipMemcpy(d_edges, G->edges, sizeof(edge) * G->m, hipMemcpyHostToDevice);
    hipMemcpy(d_dist, dist, sizeof(int) * G->n, hipMemcpyHostToDevice);
    gettimeofday(&end, NULL);
    h2d_time += (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1000000.0;

    // Run the algorithm for exactly n - 1 iterations 
    for (int i = 0; i < G->n - 1; i++) {
        h_has_next = false;

        gettimeofday(&start, NULL);
        hipMemcpy(d_has_next, &h_has_next, sizeof(bool), hipMemcpyHostToDevice);
        gettimeofday(&end, NULL);
        h2d_time += (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1000000.0;

        // Run one iteration of Bellman-Ford
        gettimeofday(&start, NULL);
        bellman_ford_one_iter<<<blocksPerGrid, threadsPerBlock>>>(G->n, G->m, d_edges, d_dist, d_has_next);
        gettimeofday(&end, NULL);
        CHECK(hipDeviceSynchronize());
        exec_time += (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1000000.0;

        gettimeofday(&start, NULL);
        hipMemcpy(&h_has_next, d_has_next, sizeof(bool), hipMemcpyDeviceToHost);
        gettimeofday(&end, NULL);
        d2h_time += (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1000000.0;
    }

    // Copy the distances back to host for negative cycle check
    gettimeofday(&start, NULL);
    hipMemcpy(dist, d_dist, sizeof(int) * G->n, hipMemcpyDeviceToHost);
    gettimeofday(&end, NULL);
    d2h_time += (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1000000.0;

    // CPU negative cycle detection
    for (int i = 0; i < G->m; i++) {
        edge e = G->edges[i];
        if (dist[e.src] < INF && dist[e.src] + e.weight < dist[e.dest]) {
            *has_negative_cycle = true;
            break;
        }
    }

    hipFree(d_edges);
    hipFree(d_dist);
    hipFree(d_has_next);

    printf("Memory allocation/setup time: %.6f seconds\n", alloc_time);
    printf("Host to Device copy time: %.6f seconds\n", h2d_time);
    printf("Device to Host copy time: %.6f seconds\n", d2h_time);
    printf("Total memory copy time: %.6f seconds\n", h2d_time + d2h_time);
    printf("CUDA execution time: %.6f seconds\n", exec_time);
}


int main(int argc, char **argv) {
    if (argc <= 2) {
        abort_with_error_message("INPUT FILE AND NUMBER OF THREADS WERE NOT FOUND!");
    }

    const char* filename = argv[1];
    int threadsPerBlock = atoi(argv[2]); // Read number of threads from command line
    if (threadsPerBlock <= 0) {
        abort_with_error_message("INVALID NUMBER OF THREADS!");
    }

    graph G;
    read_graph(filename, &G);

    int *dist = (int *)calloc(G.n, sizeof(int));
    bool has_negative_cycle = false;

    hipDeviceReset();

    int blocksPerGrid = (G.m + threadsPerBlock - 1) / threadsPerBlock;
    printf("Number of blocks: %d\n", blocksPerGrid);
    printf("Number of threads: %d\n", threadsPerBlock);

    bellman_ford(&G, dist, &has_negative_cycle, blocksPerGrid, threadsPerBlock);

    mkdir("cuda_bellman_ford_results", 0755);

    char output_filename[256];
    snprintf(output_filename, sizeof(output_filename), "cuda_bellman_ford_results/output%s", strrchr(filename, '/') + 6); // Extract file number from input filename
    write_result(output_filename, dist, G.n, has_negative_cycle);

    free(dist);
    free(G.nodes);
    free(G.edges);
    return 0;
}
